#include "GPUMem.h"

void allocGPUMem(float **d_iPyRef, float **d_dPyRef, float **d_iPyCrr, float **d_dPyCrr, float **d_kPy, float **d_ikPy, float **d_dXPy, float **d_dYPy, float **d_res, float **d_J, int lvlNum, size_t n, float **d_R, float **d_t, float **d_redArr, float **d_redArr2, float **d_A, float **d_b) {
	size_t pyLvlSize = (size_t)n;
	for (int i=0; i<lvlNum; i++) {  // TODO: allocate one big space per pyramid / for all pyramids and create a pointer Array for accessing the levels
		hipMalloc(&d_iPyRef[i], pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMalloc(&d_dPyRef[i], pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMalloc(&d_iPyCrr[i], pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMalloc(&d_dPyCrr[i], pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMalloc(&d_kPy[i], 9*sizeof(float));  CUDA_CHECK;
		hipMalloc(&d_ikPy[i], 9*sizeof(float));  CUDA_CHECK;
		hipMalloc(&d_dXPy[i], pyLvlSize*sizeof(float)); CUDA_CHECK;
		hipMalloc(&d_dYPy[i], pyLvlSize*sizeof(float)); CUDA_CHECK;
		hipMemset(d_iPyRef[i], 0, pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMemset(d_dPyRef[i], 0, pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMemset(d_iPyCrr[i], 0, pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMemset(d_dPyCrr[i], 0, pyLvlSize*sizeof(float));  CUDA_CHECK;
		hipMemset(d_kPy[i], 0, 9*sizeof(float));  CUDA_CHECK;
		hipMemset(d_ikPy[i], 0, 9*sizeof(float));  CUDA_CHECK;
		hipMemset(d_dXPy[i], 0, pyLvlSize*sizeof(float)); CUDA_CHECK;
		hipMemset(d_dYPy[i], 0, pyLvlSize*sizeof(float)); CUDA_CHECK;
		pyLvlSize = (pyLvlSize+1) / 2;
	}
	hipMalloc(d_res, n*sizeof(float));  CUDA_CHECK;
	hipMalloc(d_J, 6*n*sizeof(float));  CUDA_CHECK;
	hipMalloc(d_R, 9*sizeof(float));  CUDA_CHECK;
	hipMalloc(d_t, 3*sizeof(float));  CUDA_CHECK;
	hipMalloc(d_redArr, 27*((n+255)/256)*sizeof(float));  CUDA_CHECK;
	hipMalloc(d_redArr2, 27*((n+65535)/65536)*sizeof(float));  CUDA_CHECK;
	hipMalloc(d_A, 36*sizeof(float));  CUDA_CHECK;
	hipMalloc(d_b, 6*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_res, 0, n*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_J, 0, 6*n*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_R, 0, 9*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_t, 0, 3*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_redArr, 0, 27*((n+255)/256)*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_redArr2, 0, 27*((n+65535)/65536)*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_A, 0, 36*sizeof(float));  CUDA_CHECK;
	hipMemset(*d_b, 0, 6*sizeof(float));  CUDA_CHECK;
}

void freeGPUMem(float **d_iPyRef, float **d_dPyRef, float **d_iPyCrr, float **d_dPyCrr, float **d_kPy, float **d_ikPy, float **d_dXPy, float **d_dYPy, float *d_res, float *d_J, int lvlNum, int w, int h, float *d_R, float *d_t, float *d_redArr, float *d_redArr2, float *d_A, float *d_b) {
	size_t pyLvlSize = (size_t)w*h;
	for (int i=0; i<lvlNum; i++) {
		hipFree(d_iPyRef[i]);  CUDA_CHECK;
		hipFree(d_dPyRef[i]);  CUDA_CHECK;
		hipFree(d_iPyCrr[i]);  CUDA_CHECK;
		hipFree(d_dPyCrr[i]);  CUDA_CHECK;
		hipFree(d_kPy[i]);  CUDA_CHECK;
		hipFree(d_ikPy[i]);  CUDA_CHECK;
		hipFree(d_dXPy[i]); CUDA_CHECK;
		hipFree(d_dYPy[i]); CUDA_CHECK;
		pyLvlSize = (pyLvlSize+1) / 2;
	}
	hipFree(d_res);  CUDA_CHECK;
	hipFree(d_J);  CUDA_CHECK;
	hipFree(d_R);  CUDA_CHECK;
	hipFree(d_t);  CUDA_CHECK;
	hipFree(d_redArr);  CUDA_CHECK;
	hipFree(d_redArr2);  CUDA_CHECK;
	hipFree(d_A);  CUDA_CHECK;
	hipFree(d_b);  CUDA_CHECK;
}