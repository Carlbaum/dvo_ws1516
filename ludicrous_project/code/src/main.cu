#include "hip/hip_runtime.h"
/**
 * \file
 * \brief   This file is mostly a loop calling the tracker class, which is the one
 *          really implementing the paper.
 * \author  Oskar Carlbaum, Guillermo Gonzalez de Garibay, Georg Kuschk
 * \date    April 2016
 *
 * Program implementing the paper Robust Odometry Estimation for RGB-D Cameras
 * by Christian Kerl, Juergen Sturm, and Daniel Cremers.
 *
 * Only Gaussian and Student-T weights are implemented in this code.
 *
 * Parts of the code included in this project was contributed by Georg Kuschk
 * and the tutors of previous semesters. Parts of the code and program structure
 * have between borrowed from other student groups doing the same implementation
 * in previous semesters.
 *
 */

#include "helper.h"
#include <iostream>
#include "tum_benchmark.hpp"
#include "dataset.hpp"

//using namespace std;

// TODO: is this the proper way of using global variables inside the tracker class?
// global variables
const int MAX_LEVELS = 7;
// CUDA related
int devID;
hipDeviceProp_t props;
int g_CUDA_maxSharedMemSize;
const int g_CUDA_blockSize2DX = 16;
const int g_CUDA_blockSize2DY = 16;
const int BORDER_ZERO = 1;
const int BORDER_REPLICATE = 2;

// tracker uses these global variables, so it has to be included after them
__constant__ float const_K_pyr[9*MAX_LEVELS];     // Allocates constant memory in excess for K and K downscaled. Stored column-wise and matrix after matrix
__constant__ float const_RK_inv[9];     // Allocates space for the concatenation of a rotation and an intrinsic matrix. Stored column-wise
__constant__ float const_translation[3];     // Allocates space for a translation vector
texture <float, 2, hipReadModeElementType> texRef_grayImg;
texture <float, 2, hipReadModeElementType> texRef_gray_dx;
texture <float, 2, hipReadModeElementType> texRef_gray_dy;

#include "tracker.hpp"

int main(int argc, char *argv[]) {
        //_______________________________________________________
        //_______________________________________________________
        //________ SET ENVIRONMENT
        //_______________________________________________________
        //_______________________________________________________

        std::cout << std::endl
                  << "*******************************************************\n"
                  << "*********DENSE VISUAL ODOMETRY PROGRAM STARTED*********\n"
                  << "*******************************************************\n" << std::endl;

#ifdef ENABLE_CUBLAS
        std::cout << "Using cuBLAS" << std::endl;
#endif

        // ---------- PARAMETERS ----------
        // these can be give through command line:

        // Path to data set
        // this program will use all the images described in the txt files
        // e.g. "-path ../data/mypath_to_dataset"
        std::string path = "../data/freiburg1_xyz_first_10";
        getParam("path", path, argc, argv);
        std::cout << "Path to dataset: " << path << std::endl;

        // gives the number of levels of the pyramids,
        // This number cannot be arbitrary and will be checked later.
        // e.g. "-numberOfLevels 4"
        int numberOfLevels = 5;
        getParam("numberOfLevels", numberOfLevels, argc, argv);
        numberOfLevels = std::max(1, numberOfLevels);
        numberOfLevels = std::min(MAX_LEVELS, numberOfLevels); // 1/512 size reduction is in some cases already too large

        // set to true to use Student-T weights.
        // e.g. "-tDistWeights 1" for true
        bool tDistWeights = false;
        getParam("tDistWeights", tDistWeights, argc, argv);
        std::cout << "tDistWeights: " << tDistWeights << std::endl;

        // ------- END OF PARAMETERS -------


        // Dataset instantiation
        Dataset dataset(path);
        std::vector<double> timestamps;
        Eigen::Matrix3f K = dataset.K;

        // These poses will eventually be printed as output
        std::vector<Eigen::Matrix4f> poses;

        // Load images for frame 0, for initialization purposes
        cv::Mat mGray = loadIntensity(dataset.frames[0].colorPath);
        cv::Mat mDepth = loadDepth(dataset.frames[0].depthPath);

        // get image dimensions
        int w = mGray.cols;
        int h = mGray.rows;

        // Determine number of downscaling levels based on the size of input images
        // Compute the number of scale levels, based on the image size, the
        // pyramid scale factor and the minimum image size at the coarsest level
        const int MIN_IMAGE_SIZE = 32;
        int m_downscaleFactor = 2;
        int m_nLevels = 1;
        int w_tmp = w;
        int h_tmp = h;
        while( w_tmp/m_downscaleFactor >= MIN_IMAGE_SIZE
               && h_tmp/m_downscaleFactor >= MIN_IMAGE_SIZE ) {
                w_tmp = (int) (w_tmp / m_downscaleFactor);
                h_tmp = (int) (h_tmp / m_downscaleFactor);
                m_nLevels++;
        }
        numberOfLevels = std::max(1, std::min(m_nLevels, numberOfLevels));
        std::cout << "number of levels in pyramids: " << numberOfLevels << std::endl;

        // allocate raw input intensity and depth arrays
        float *imgGray = new float[(size_t)w*h];
        float *imgDepth = new float[(size_t)w*h];

        //_______________________________________________________
        //_______________________________________________________
        //________ EXECUTION
        //_______________________________________________________
        //_______________________________________________________

        // convert opencv images to arrays
        convert_mat_to_layered(imgGray, mGray);
        convert_mat_to_layered(imgDepth, mDepth);

        // initialize the tracker
        Tracker tracker(imgGray, imgDepth, w, h, K, 0, numberOfLevels-1,tDistWeights);

        // Store pose for frame 0
        poses.push_back(Matrix4f::Identity());
        timestamps.push_back(dataset.frames[0].timestamp);

        float total_time = 0.0f;
        std::cout << "\nStarting main loop, reading images and calculating trajectory. Take a chill pill, this may take a while!\n" << std::endl;

        // main loop
        Vector6f xi_current;
        for (size_t i = 1; i < dataset.frames.size(); ++i) {
                Timer timer; timer.start();

                // Load in the images of the next frame
                mGray = loadIntensity(dataset.frames[i].colorPath);
                mDepth = loadDepth(dataset.frames[i].depthPath);

                // convert opencv images to arrays
                convert_mat_to_layered(imgGray, mGray);
                convert_mat_to_layered(imgDepth, mDepth);

                // std::cout << "Image number: " << i << std::endl;
                xi_current = tracker.align(imgGray, imgDepth);

                timer.end();  float t = 1000 * timer.get(); // elapsed time in seconds
                total_time += t;
                // std::cout << "Time of loading + doing calculations on image #" << i << ": " << t << " ms" << std::endl;
                // show input image
                // showImage("Input " + std::to_string(i), mGray, 100+20*i, 100+10*i);  // show at position (x_from_left=100,y_from_above=100)

                // Update and push absolute pose
                poses.push_back(lieExp(xi_current));
                timestamps.push_back(dataset.frames[i].timestamp);
        }

        //_______________________________________________________
        //_______________________________________________________
        //________ OUTPUT
        //_______________________________________________________
        //_______________________________________________________

        // Save poses to disk
        std::cout << std::endl  << "Total time for loading + doing calculations on "
                  << dataset.frames.size() << " images took " << total_time
                  << " ms.\nThis gives us an average of "
                  << total_time/dataset.frames.size()
                  << " ms per frame.\n" << std::endl;

        std::string options = "";
        if (tDistWeights) {
                options += "_tdist";
        } else {
                options += "_gdist";
        }
    #ifdef ENABLE_CUBLAS
        options += "_cublas";
    #else
        options += "_nocublas";
    #endif

        savePoses( path +options+ "_trajectory.txt", poses, timestamps);

        //_______________________________________________________
        //_______________________________________________________
        //________ CLOSE
        //_______________________________________________________
        //_______________________________________________________

        cv::waitKey(0);
        cvDestroyAllWindows();
        std::cout << "All done! Check out the output file: " << path << options << "_trajectory.txt for the resulting trajectory!\n" << std::endl;
        return 0;
}
