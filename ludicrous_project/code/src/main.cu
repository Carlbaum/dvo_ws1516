#include "hip/hip_runtime.h"
/**
 * Some documentations should probably go here ;)
 * Dense visual odometry somethingsomething
 */

#include "helper.h"
#include <iostream>
#include "tum_benchmark.hpp"
#include "dataset.hpp"

using namespace std;

// TODO: is this the proper way of using global variables inside the tracker class?
    // global variables
    const int MAX_LEVELS = 10;
        // CUDA related
    int             devID;
    hipDeviceProp_t  props;
    int g_CUDA_maxSharedMemSize;
    const int g_CUDA_blockSize2DX = 16;
    const int g_CUDA_blockSize2DY = 16;
    const int BORDER_ZERO = 1;
    const int BORDER_REPLICATE = 2;
    // tracker uses these global variables, so it has to be included after them
    __constant__ float const_K_pyr[9*MAX_LEVELS]; // Allocates constant memory in excess for K and K downscaled
    __constant__ float const_RK_inv[9]; // Allocates space for the concatenation of a rotation and an intrinsic matrix
    __constant__ float const_translation[3]; // Allocates space for a translation vector
#include "tracker.hpp"

int main(int argc, char *argv[]) {

    hipDeviceSynchronize();  CUDA_CHECK;

    // Get information about the GPU
    hipGetDevice(&devID); CUDA_CHECK;
    hipGetDeviceProperties(&props, devID); CUDA_CHECK;
    g_CUDA_maxSharedMemSize = props.sharedMemPerBlock;



    // ---------- PARAMETERS ----------

    // Path to data set
    // this program will use all the images described in the txt files
    std::string path = "../data/freiburg1_xyz_first_10";
    getParam("path", path, argc, argv);
    std::cout << "Path to dataset: " << path << std::endl;

    // gives the number of levels of the pyramids
    int numberOfLevels = 5;
    getParam("numberOfLevels", numberOfLevels, argc, argv);
    numberOfLevels = std::max(1, numberOfLevels);
    numberOfLevels = std::min(MAX_LEVELS, numberOfLevels); // 1/512 size reduction is in some cases already too large
    std::cout << "number of levels in pyramids: " << numberOfLevels << std::endl;

    /* FROM THE EXERCISES, DON'T THINK WE NEED THIS
        // input image
        string image = "";
        bool ret = getParam("i", image, argc, argv);
        if (!ret) cerr << "ERROR: no image specified" << endl;
        if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
    */
    // ------- END OF PARAMETERS -------


    // Dataset declarations
    Dataset dataset(path);
    std::vector<double> timestamps;
    Eigen::Matrix3f K = dataset.K;

    // This will eventually be printed as output
    //std::vector<Eigen::Matrix4f> poses;

    // Load images for frame 0, for initialization purposes
    cv::Mat mGray = loadIntensity(dataset.frames[0].colorPath);
    cv::Mat mDepth = loadDepth(dataset.frames[0].depthPath);

    // get image dimensions
    int w = mGray.cols;
    int h = mGray.rows;

    // allocate raw input intensity and depth arrays
    float *imgGray = new float[(size_t)w*h];
    float *imgDepth = new float[(size_t)w*h];

    // convert opencv images to arrays
    convert_mat_to_layered(imgGray, mGray);
    convert_mat_to_layered(imgDepth, mDepth);

    // initialize the tracker
    Tracker tracker(imgGray, imgDepth, w, h, K, 0, numberOfLevels-1);

    // TODO: WE NEED TO INITIALIZE THE IMAGES BEFORE THE MAIN LOOP

    /* This is not really needed yet, uncomment when we need it
    // Store pose for frame 0
    poses.push_back(Matrix4f::Identity());
    timestamps.push_back(dataset.frames[0].timestamp);
    */

    std::cout << "Hello world" << std::endl;
    for (size_t i = 1; i < dataset.frames.size(); ++i) {
        Timer timer; timer.start();

        // Load in the images of the next frame
        mGray = loadIntensity(dataset.frames[i].colorPath);
        mDepth = loadDepth(dataset.frames[i].depthPath);

        // convert opencv images to arrays
        convert_mat_to_layered(imgGray, mGray);
        convert_mat_to_layered(imgDepth, mDepth);

        // TODO: THIS IS WHERE WE SHOULD CALL THE ALIGN FUNCITON
        tracker.align(imgGray, imgDepth);

        timer.end();  float t = timer.get();  // elapsed time in seconds

        cout << "Time of loading + doing calculations on image #" << i  << ": " << t*1000 << " ms" << endl;
        // show input image
        // showImage("Input " + std::to_string(i), mGray, 100+20*i, 100+10*i);  // show at position (x_from_left=100,y_from_above=100)
    }


    cv::waitKey(0);
    cvDestroyAllWindows();
    std::cout << "Goodbye world" << std::endl;
    return 0;
}
